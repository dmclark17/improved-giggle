#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "data_manager.h"

#include "gpu_benchmark.h"

void cublass_gemm_execute(GemmRun* run) {

    hipblasOperation_t transa, transb;

    transa = HIPBLAS_OP_T;
    transb = HIPBLAS_OP_T;

    size_t pitch_A, pitch_B, pitch_C;
    float* cuda_A;
    float* cuda_B;
    float* cuda_C;

    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipError_t err;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return;
    }

    hipMallocPitch(&cuda_A, &pitch_A, run->k * sizeof(float), run->m);
    hipMallocPitch(&cuda_B, &pitch_B, run->n * sizeof(float), run->k);
    hipMallocPitch(&cuda_C, &pitch_C, run->n * sizeof(float), run->m);
    if ((err = hipGetLastError()) != hipSuccess) {
        printf("%s\n", hipGetErrorString(err));
        return;
    }

    stat = hipblasSetMatrix(run->m, run->k, sizeof(float), run->a, run->lda, cuda_A, pitch_A / sizeof(float));
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Set A matrix failed\n");
        return;
    }

    stat = hipblasSetMatrix(run->k, run->n, sizeof(float), run->b, run->ldb, cuda_B, pitch_B / sizeof(float));
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Set B matrix failed\n");
        return;
    }


    stat = hipblasSgemm(handle, transa, transb,
                       run->m, run->n, run->k,
                       &(run->alpha),
                       cuda_A, pitch_A / sizeof(float),
                       cuda_B, pitch_B / sizeof(float), &(run->beta),
                       cuda_C, pitch_C / sizeof(float));
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Multiplication failed\n");
        return;
    }

    hipblasGetMatrix(run->m, run->n, sizeof(float), cuda_C, pitch_C / sizeof(float),
                    run->c, run->ldc);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Get matrix failed\n");
        return;
    }

    hipFree(cuda_A);
    hipFree(cuda_B);
    hipFree(cuda_C);
    // std::cout << "Pitch " << pitch_A << " size " << run->k * sizeof(float) << std::endl;
}
