#include <cstdio>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "data_manager.h"

#include "gpu_util.h"

void init_cuda_matrices(GemmRun<float>* run,
                        size_t* pitch_A, size_t* pitch_B, size_t* pitch_C,
                        float** cuda_A, float** cuda_B, float** cuda_C) {

    hipblasStatus_t stat;
    hipError_t err;

    hipMallocPitch(cuda_A, pitch_A, run->k * sizeof(float), run->m);
    hipMallocPitch(cuda_B, pitch_B, run->n * sizeof(float), run->k);
    hipMallocPitch(cuda_C, pitch_C, run->n * sizeof(float), run->m);
    if ((err = hipGetLastError()) != hipSuccess) {
        printf("%s\n", hipGetErrorString(err));
        return;
    }


    stat = hipblasSetMatrix(run->m, run->k, sizeof(float), run->a, run->lda, *cuda_A, *pitch_A / sizeof(float));
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Set A matrix failed\n");
        return;
    }

    stat = hipblasSetMatrix(run->k, run->n, sizeof(float), run->b, run->ldb, *cuda_B, *pitch_B / sizeof(float));
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Set B matrix failed\n");
        return;
    }
}


void deinit_cuda_matrices(GemmRun<float>* run, size_t pitch_C,
                          float* cuda_A, float* cuda_B, float* cuda_C) {

    hipblasStatus_t stat;

    stat = hipblasGetMatrix(run->m, run->n, sizeof(float), cuda_C, pitch_C / sizeof(float),
              run->c, run->ldc);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Get matrix failed\n");
        return;
    }

    hipFree(cuda_A);
    hipFree(cuda_B);
    hipFree(cuda_C);
}
